#include "hip/hip_runtime.h"
#include "../gradient.h"
#include "../timing.c"

#define TILE_WIDTH   (CONV_BLOCK_SIDE + MASK_SIZE - 1)


__constant__ int sobelX[MASK_SIZE*MASK_SIZE*sizeof(int)];
__constant__ int sobelY[MASK_SIZE*MASK_SIZE*sizeof(int)];



__global__ void convolutions_gpu(unsigned char *input_image, unsigned char *img_grad_x, unsigned char *img_grad_v, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int radius = MASK_RADIUS;
    int block_m_radius = CONV_BLOCK_SIDE - radius;

    __shared__ unsigned char img_shared[TILE_WIDTH][TILE_WIDTH];

    // Top side of the block
    if ((threadIdx.y < radius) ) {

        // top left corner of the block
        if (threadIdx.x < radius && (x-radius) >= 0 && (y-radius) >= 0)
            img_shared[threadIdx.y][threadIdx.x] = input_image[(y-radius) * width + x - radius];
    
        // top right corner of the block
        if (threadIdx.x >= block_m_radius && (x+radius) < width && (y-radius) >= 0) 
            img_shared[threadIdx.y][threadIdx.x + 2*radius] = input_image[(y-radius) * width + x + radius];
        
        // top side of the block
        if ((y-radius) >= 0) 
            img_shared[threadIdx.y][threadIdx.x + radius] = input_image[(y-radius) * width + x];  
    }

    // Bottom side of the block
    if (threadIdx.y >= block_m_radius) {
    
        // bottom left corner of the block
        if (threadIdx.x < radius && (x-radius) >= 0 && (y+radius) < height)
            img_shared[threadIdx.y + 2*radius][threadIdx.x] = input_image[(y+radius) * width + x - radius];

        // bottom right corner of the block
        if (threadIdx.x >= block_m_radius && (y+radius) < height) 
            img_shared[threadIdx.y + 2*radius][threadIdx.x + 2*radius] = input_image[(y+radius) * width + x + radius];
    
        // bottom side of the block
        if ((y+radius) < height) 
            img_shared[threadIdx.y + 2*radius][threadIdx.x + radius] = input_image[(y+radius) * width + x];  
    }

    // Left side of the block
    if (threadIdx.x < radius) {
        if ((x-radius) >= 0) {
            img_shared[threadIdx.y + radius][threadIdx.x] = input_image[y * width + x - radius];  
        }
    }

    // Right side of the block
    if (threadIdx.x >= block_m_radius) {
        if ((x+radius) < width) {
            img_shared[threadIdx.y + radius][threadIdx.x + 2*radius] = input_image[y * width + x + radius];  
        }
    }
      
    // center of the block
	img_shared[radius + threadIdx.y][radius + threadIdx.x] = input_image[y * width + x];
	
    // END SHARED MEMORY LOADING
	__syncthreads();
    
    int sum_x = 0;
    int sum_y = 0;
	for (int i = 0; i < MASK_SIZE; i++) {
		for (int j = 0; j < MASK_SIZE; j++) {
            sum_x += img_shared[threadIdx.y + i][threadIdx.x + j] * sobelX[i*MASK_SIZE + j];
            sum_y += img_shared[threadIdx.y + i][threadIdx.x + j] * sobelY[i*MASK_SIZE + j];
        }
    }
	
    __syncthreads();
    
    // write in global memory
    img_grad_x[y*width + x] = abs(sum_x);
    img_grad_v[y*width + x] = abs(sum_y);
}



void cuda_compute_gradients(unsigned char *img_gray, unsigned char *img_grad_h, unsigned char *img_grad_v, int width, int height, int num_streams, char *log_file) {
    unsigned char *d_img_gray, *d_grad_h, *d_grad_v;

    size_t size = width*height;

    const int h_sobelX[] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
    const int h_sobelY[] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};
    const size_t mask_dim = sizeof(int)*MASK_SIZE*MASK_SIZE;

    CHECK(hipMalloc((void **)&d_img_gray, size));
    CHECK(hipMalloc((void **)&d_grad_h, size));
    CHECK(hipMalloc((void **)&d_grad_v, size));
    if(d_img_gray == NULL || d_grad_h == NULL || d_grad_v == NULL) {
        printf("Unable to allocate memory on GPU.\n");
        exit(EXIT_FAILURE);
    }

    dim3 block(CONV_BLOCK_SIDE, CONV_BLOCK_SIDE);
    dim3 grid((width + CONV_BLOCK_SIDE - 1)/CONV_BLOCK_SIDE, (height + CONV_BLOCK_SIDE - 1)/CONV_BLOCK_SIDE);

    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));
    float time;

    if(num_streams>1) {
        while((size % num_streams) != 0)
            num_streams++;
        
        int stream_size = size/num_streams;
        grid.x = (stream_size + block.x - 1)/block.x;
        
        hipStream_t streams[num_streams];
        for(int idx=0; idx<num_streams; idx++) {
            CHECK(hipStreamCreateWithFlags(&streams[idx], hipStreamNonBlocking));
        }

        // Pinned memory allocation
        unsigned char *img_gray_pnd, *img_grad_h_pnd, *img_grad_v_pnd;
        int stream_idx = 0;
        CHECK(hipHostAlloc((void **)&img_gray_pnd, size, hipHostMallocDefault));
        CHECK(hipHostAlloc((void **)&img_grad_h_pnd, size, hipHostMallocDefault));
        CHECK(hipHostAlloc((void **)&img_grad_v_pnd, size, hipHostMallocDefault));

        CHECK(hipEventRecord(start, 0));
        CHECK(hipMemcpy(img_gray_pnd, img_gray, size, hipMemcpyHostToHost));
        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(sobelX), &h_sobelX, mask_dim));
        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(sobelY), &h_sobelY, mask_dim));

        for(int idx=0; idx<num_streams; idx++) {
            stream_idx = idx * stream_size;
            CHECK(hipMemcpyAsync(&d_img_gray[stream_idx], &img_gray_pnd[stream_idx], stream_size, hipMemcpyHostToDevice, streams[idx]));
            convolutions_gpu<<<grid, block, 0, streams[idx]>>>(&d_img_gray[stream_idx], &d_grad_h[stream_idx], &d_grad_v[stream_idx], width, height);
            CHECK(hipMemcpyAsync(&img_grad_h_pnd[stream_idx], &d_grad_h[stream_idx], stream_size, hipMemcpyDeviceToHost, streams[idx]));
            CHECK(hipMemcpyAsync(&img_grad_v_pnd[stream_idx], &d_grad_v[stream_idx], stream_size, hipMemcpyDeviceToHost, streams[idx]));
        }
        CHECK(hipDeviceSynchronize());
        CHECK(hipMemcpy(img_grad_h, img_grad_h_pnd, size, hipMemcpyHostToHost));
        CHECK(hipMemcpy(img_grad_v, img_grad_v_pnd, size, hipMemcpyHostToHost));
        CHECK(hipEventRecord(end, 0));

        // Free some memory
        CHECK(hipHostFree(img_gray_pnd));
        CHECK(hipHostFree(img_grad_h_pnd));
        CHECK(hipHostFree(img_grad_v_pnd));

        // Destroy streams
        for(int idx=0; idx<num_streams; idx++) {
            CHECK(hipStreamDestroy(streams[idx]));
        }
    }

    else {
        // Data transfer H2D
        CHECK(hipEventRecord(start, 0));
        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(sobelX), &h_sobelX, mask_dim));
        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(sobelY), &h_sobelY, mask_dim));
        CHECK(hipMemcpy(d_img_gray, img_gray, size, hipMemcpyHostToDevice));
        
        convolutions_gpu<<<grid, block>>>(d_img_gray, d_grad_h, d_grad_v, width, height);
        CHECK(hipDeviceSynchronize());
        
        // D2H transfer
        CHECK(hipMemcpy(img_grad_h, d_grad_h, size, hipMemcpyDeviceToHost));
        CHECK(hipMemcpy(img_grad_v, d_grad_v, size, hipMemcpyDeviceToHost));
        CHECK(hipEventRecord(end, 0));
    }

    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    time /= 1000;
    printf("[Gradients] - GPU Elapsed time: %f sec\n\n", time);
    //write_to_file(log_file, "Gradients", time, 1, 0);                     // Generates Buffer Overflow in colab
    
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        printf("\n--> Error: %s\n", hipGetErrorString(err));
    }

    CHECK(hipFree(d_img_gray));
    CHECK(hipFree(d_grad_h));
    CHECK(hipFree(d_grad_v));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end));
}

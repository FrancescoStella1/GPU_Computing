#include "hip/hip_runtime.h"
#include "../grayscale.h"
#include "../common.h"
#include "../timing.c"

#define BLOCKDIM   32


__global__ void grayscale_gpu(unsigned char *img, unsigned char *img_gray, const size_t size) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size)
        return;
    
    uint idx = i*3;
    unsigned char r, g, b;
    r = img[idx];
    g = img[idx+1];
    b = img[idx+2];
    
    // grayscale conversion
    img_gray[i] = ((0.299*r) + (0.587*g) + (0.114*b));     // from it.mathworks.com - rgb2gray
}



void cuda_convert(unsigned char *h_img, unsigned char *h_img_gray, int width, int height, char *log_file) {
    // Device memory allocation
    unsigned char *d_img;
    unsigned char *d_img_gray;
    const size_t size = width*height;

    CHECK(hipMalloc((void **)&d_img, size*3));   // 3 channels
    CHECK(hipMalloc((void **)&d_img_gray, size));  
    if(d_img == NULL || d_img_gray == NULL)   {
        printf("Unable to allocate memory on GPU.\n");
        exit(EXIT_FAILURE);
    }

    // Data transfer H2D
    CHECK(hipMemcpy(d_img, h_img, size*3, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_img_gray, h_img_gray, size, hipMemcpyHostToDevice));

    // Kernel launch
    dim3 block(BLOCKDIM);
    dim3 grid((size+block.x-1)/block.x);

    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));
    float time;
    hipEventRecord(start, 0);
    grayscale_gpu<<< grid, block >>>(d_img, d_img_gray, size);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("[Grayscale] - GPU Elapsed time: %f sec\n\n", time/1000);
    write_to_file(log_file, "Grayscale", time/1000, 1, 0);

    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        printf("\n--> Error: %s\n", hipGetErrorString(err));
    }

    // Data transfer H2D
    CHECK(hipMemcpy(h_img_gray, d_img_gray, size, hipMemcpyDeviceToHost));

    // Free memory
    CHECK(hipFree(d_img));
    CHECK(hipFree(d_img_gray));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end));
}
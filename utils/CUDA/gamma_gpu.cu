#include "hip/hip_runtime.h"
#include "../gamma.h"
#include "../common.h"

#define BLOCKDIM   32


__global__ void gamma_correction_gpu(struct Histogram *hist, unsigned char *img_gray, const size_t size) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i>=size)
        return;

    //atomicAdd(&(hist->num[img_gray[i]/L]), 1);      // provide L
    
}


void cuda_gamma_correction(struct Histogram *hist, unsigned char *h_img_gray, const size_t size) {
    unsigned char *d_img_gray;
    size_t nBytes = sizeof(struct Histogram);
    struct Histogram *d_hist;

    // Device memory allocation
    CHECK(hipMalloc((void **)&d_hist, nBytes));
    CHECK(hipMalloc((void **)&d_img_gray, size));
    if(d_img_gray == NULL) {
        printf("Unable to allocate memory on GPU.\n");
        exit(EXIT_FAILURE);
    }

    // Data transfer H2D
    CHECK(hipMemcpy(d_hist, hist, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_img_gray, h_img_gray, size, hipMemcpyHostToDevice));

    // Kernel launch
    dim3 block;
    dim3 grid;
    block.x = BLOCKDIM;
    grid.x = ((size + block.x - 1)/block.x);

    gamma_correction_gpu<<< grid, block >>>(hist, d_img_gray, size);
    CHECK(hipDeviceSynchronize());

    // Data transfer D2H
    CHECK(hipMemcpy(hist, d_hist, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_img_gray, d_img_gray, size, hipMemcpyDeviceToHost));

    // Free memory
    CHECK(hipFree(d_hist));
    CHECK(hipFree(d_img_gray));
}
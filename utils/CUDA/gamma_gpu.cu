#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../gamma.h"
#include "../common.h"

#define BLOCKDIM   32


__global__ void create_hist_gpu(unsigned int *num, unsigned char *img_gray, const size_t size) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i>=size)
        return;
        
    atomicAdd((unsigned int *)&num[(img_gray[i]/L)], 1);
}


__global__ void apply_gamma_gpu(unsigned char *img_gray, double gamma, double factor, const size_t size) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i>=size)
      return;
    
    img_gray[i] = (unsigned char)(factor*pow(img_gray[i], 1/gamma));

}


void cuda_gamma_correction(unsigned char *h_img_gray, const size_t size) {
    struct Histogram *hist = createHistogram();
    unsigned char *max_intensity = (unsigned char *)calloc(1, sizeof(unsigned char));
    size_t nBytes = (256/L)*sizeof(unsigned int);
    unsigned char *d_img_gray;
    unsigned int *d_num;
    double g = 0;
    double factor = 0;

    // Device memory allocation
    CHECK(hipMalloc((void **)&d_num, nBytes));
    CHECK(hipMalloc((void **)&d_img_gray, size));
    if(d_num == NULL || d_img_gray == NULL) {
        printf("Unable to allocate memory on GPU.\n");
        exit(EXIT_FAILURE);
    }

    
    dim3 block(BLOCKDIM);
    dim3 grid((size + block.x - 1)/block.x);
    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));
    float time;

    // Data transfer H2D
    CHECK(hipMemcpy(d_num, hist->num, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_img_gray, h_img_gray, size, hipMemcpyHostToDevice));

    // Run kernel
    hipEventRecord(start, 0);
    create_hist_gpu<<< grid, block >>>(d_num, d_img_gray, size);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("GPU Elapsed time: %f sec\n\n", time/1000);
    
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        printf("\n--> Error: %s\n", hipGetErrorString(err));
    }

    // Data transfer D2H
    CHECK(hipMemcpy(hist->num, d_num, nBytes, hipMemcpyDeviceToHost));
    //CHECK(hipMemcpy(h_img_gray, d_img_gray, size, hipMemcpyDeviceToHost));

    // Free memory
    CHECK(hipFree(d_num));


    // Compute cumulative histogram and normalized gamma value on CPU
    g = compute_gamma(hist->num, hist->cnum, size, max_intensity);
    printf("Maximum pixel intensity in the grayscale image: %u\n", *max_intensity);
    factor = *max_intensity/pow(*max_intensity, 1/g);
    printf("Normalized gamma value: %f\n", g);
    printf("Factor: %f\n", factor);
    printf("Max intensity: %u\n", *max_intensity);
    
    // Run second kernel
    hipEventRecord(start, 0);
    apply_gamma_gpu<<< grid, block >>>(d_img_gray, g, factor, size);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("GPU Elapsed time: %f sec\n\n", time/1000);

    err = hipGetLastError();
    if(err != hipSuccess) {
        printf("\n--> Error: %s\n", hipGetErrorString(err));
    }

    // Data transfer D2H
    CHECK(hipMemcpy(h_img_gray, d_img_gray, size, hipMemcpyDeviceToHost));

    // Free memory
    CHECK(hipFree(d_img_gray));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end));
    
}
#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../gamma.h"
#include "../common.h"

#define BLOCKDIM   32


__global__ void old_create_hist_gpu(unsigned int *num, unsigned char *img_gray, const size_t size) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i>=size)
        return;
        
    atomicAdd((unsigned int *)&num[(img_gray[i]/L)], 1);
}

__global__ void create_hist_gpu(unsigned int *num, unsigned char *img_gray, unsigned int *max_intensity, const size_t size) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i>=size)
        return;

    // Create array s_num in order to store *num in shared memory
    __shared__ unsigned int s_num[256/L];
    s_num[threadIdx.x] = 0;
    if(threadIdx.x >= 16) {
      s_num[(threadIdx.x*2)] = 0;
      s_num[(threadIdx.x*2)+1] = 0;
    }

    __syncthreads();
    
    // Variable in shared memory to find block-level maximum intensity
    __shared__ unsigned int s_intensity;
    if(threadIdx.x==0)
      s_intensity = 0;
    
    __syncthreads();

    unsigned int intensity = (unsigned int)img_gray[i];
    atomicAdd((unsigned int *)&s_num[(intensity/L)], 1);
    __syncthreads();
    
    for(int idx=256/L; idx>0; idx--) {
        if(s_num[idx] > 0 and (intensity/L) < idx)                // if there are intensities greater than the intensity of this thread, break the loop
          break;
        if(s_num[idx] > 0 and (intensity/L) == idx) {             // otherwise, compute the maximum intensity in the block
          atomicMax((unsigned int *)&s_intensity, intensity);
          break;
        }
    }
    __syncthreads();

    atomicAdd((unsigned int *)&num[threadIdx.x], s_num[threadIdx.x]);
    if(threadIdx.x >= 16) {
      atomicAdd((unsigned int *)&num[(threadIdx.x*2)], s_num[(threadIdx.x*2)]);
      atomicAdd((unsigned int *)&num[(threadIdx.x*2)+1], s_num[(threadIdx.x*2)+1]);
    }
    __syncthreads();

    if(threadIdx.x == 0) {
      atomicMax((unsigned int *)max_intensity, s_intensity);
    }
    __syncthreads();
}


__global__ void apply_gamma_gpu(unsigned char *img_gray, double gamma, double factor, const size_t size) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i>=size)
      return;
    
    img_gray[i] = (unsigned char)(factor*pow(img_gray[i], 1/gamma));

}


void cuda_gamma_correction(unsigned char *h_img_gray, const size_t size, char *log_file) {
    struct Histogram *hist = createHistogram();
    for(int idx=0; idx < (256/L); idx++) {
        hist->num[idx] = 0;
        hist->cnum[idx] = 0;
    }
    unsigned int *h_max_intensity = (unsigned int *)malloc(sizeof(unsigned int));
    *h_max_intensity = 0;
    size_t nBytes = (256/L)*sizeof(unsigned int);
    size_t nBytes_1 = sizeof(unsigned int);
    unsigned char *d_img_gray;
    unsigned int *d_num;
    unsigned int *d_max_intensity;
    double g = 0;
    double factor = 0;

    // Device memory allocation
    CHECK(hipMalloc((unsigned int **)&d_num, nBytes));
    CHECK(hipMalloc((unsigned char **)&d_img_gray, size));
    CHECK(hipMalloc((unsigned int **)&d_max_intensity, sizeof(unsigned int)));
    if(d_num == NULL || d_img_gray == NULL || d_max_intensity == NULL) {
        printf("Unable to allocate memory on GPU.\n");
        exit(EXIT_FAILURE);
    }

    
    dim3 block(BLOCKDIM);
    dim3 grid((size + block.x - 1)/block.x);
    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));
    float time;

    // Data transfer H2D
    CHECK(hipMemcpy(d_num, hist->num, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_img_gray, h_img_gray, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_max_intensity, h_max_intensity, nBytes_1, hipMemcpyHostToDevice));

    // Run kernel
    hipEventRecord(start, 0);
    create_hist_gpu<<< grid, block >>>(d_num, d_img_gray, d_max_intensity, size);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("GPU Elapsed time: %f sec\n\n", time/1000);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        printf("\n--> Error: %s\n", hipGetErrorString(err));
    }

    // Data transfer D2H
    CHECK(hipMemcpy(hist->num, d_num, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_img_gray, d_img_gray, size, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_max_intensity, d_max_intensity, nBytes_1, hipMemcpyDeviceToHost));
    CHECK(hipDeviceSynchronize());

    // Free memory
    CHECK(hipFree(d_num));
    CHECK(hipFree(d_max_intensity));


    // Compute cumulative histogram and normalized gamma value on CPU
    g = compute_gamma(hist->num, hist->cnum, size);
    factor = *h_max_intensity/pow(*h_max_intensity, 1/g);
    printf("Normalized gamma value: %f\n", g);
    printf("Factor: %f\n", factor);
    printf("Max intensity: %u\n", *h_max_intensity);
    
    // Run second kernel
    hipEventRecord(start, 0);
    apply_gamma_gpu<<< grid, block >>>(d_img_gray, g, factor, size);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("GPU Elapsed time: %f sec\n\n", time/1000);

    err = hipGetLastError();
    if(err != hipSuccess) {
        printf("\n--> Error: %s\n", hipGetErrorString(err));
    }

    // Data transfer D2H
    CHECK(hipMemcpy(h_img_gray, d_img_gray, size, hipMemcpyDeviceToHost));

    // Free memory
    CHECK(hipFree(d_img_gray));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end));
    
}
#include "hip/hip_runtime.h"
#include "../hog_utils.h"


__global__ void mag_dir_gpu(unsigned char *gradientX, unsigned char *gradientY, unsigned char *magnitude, unsigned char *direction, int size) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size)
        return;
    
    float grad_x = gradientX[i];
    float grad_y = gradientY[i];

    float mag = sqrtf(powf(grad_x, 2) + powf(grad_y, 2));
    float atang = atan2f(grad_y, grad_x) * (180/PI);

    magnitude[i] = (unsigned char)mag;
    direction[i] = (unsigned char)atang;

}


__global__ void hog_gpu(float *bins, unsigned char *magnitude, unsigned char *direction, int width, int height) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    uint j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i >= width || j >= height)
        return;

    int lbin = direction[i*width + j]/DELTA_THETA;
    int ubin = lbin + 1;
    if(ubin>=NUM_BINS)
      ubin = 0;

    int cbin = (lbin + 0.5);

    float l_value = magnitude[i*width + j] * ((direction[i*width + j] - DELTA_THETA/2)/DELTA_THETA);  // value of the j-th bin
    float u_value = magnitude[i*width + j] * ((direction[i*width + j] - cbin)/DELTA_THETA);

    int blocks_per_row = (width + HOG_BLOCK_SIDE - 1)/HOG_BLOCK_SIDE;
    int block_idx = blockIdx.y * blocks_per_row + blockIdx.x;
    atomicAdd(&bins[block_idx*NUM_BINS + lbin], l_value);
    atomicAdd(&bins[block_idx*NUM_BINS + ubin], u_value);
}


void cuda_compute_mag_dir(unsigned char *gradientX, unsigned char *gradientY, unsigned char *magnitude, unsigned char *direction, int dim) {

    unsigned char *d_gradientX;
    unsigned char *d_gradientY;
    unsigned char *d_magnitude;
    unsigned char *d_direction;
    size_t size = dim;

    CHECK(hipHostMalloc((void **)&d_gradientX, size));
    CHECK(hipHostMalloc((void **)&d_gradientY, size));
    CHECK(hipHostMalloc((void **)&d_magnitude, size));
    CHECK(hipHostMalloc((void **)&d_direction, size));

    if(d_gradientX == NULL || d_gradientY == NULL || d_magnitude == NULL || d_direction == NULL)   {
        printf("Unable to allocate memory on GPU.\n");
        exit(EXIT_FAILURE);
    }

    CHECK(hipMemcpyAsync(d_gradientX, gradientX, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpyAsync(d_gradientY, gradientY, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpyAsync(d_magnitude, magnitude, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpyAsync(d_direction, direction, size, hipMemcpyHostToDevice));

    CHECK(hipDeviceSynchronize());

    dim3 block(HOG_BLOCK_SIDE);
    dim3 grid((size+block.x-1)/block.x);

    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));
    float time;
    
    hipEventRecord(start, 0);
    mag_dir_gpu<<< grid, block >>>(d_gradientX, d_gradientY, d_magnitude, d_direction, dim);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("GPU Elapsed time: %f sec\n\n", time/1000);

    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        printf("\n--> Error: %s\n", hipGetErrorString(err));
    }

    CHECK(hipMemcpyAsync(magnitude, d_magnitude, size, hipMemcpyDeviceToHost));
    CHECK(hipMemcpyAsync(direction, d_direction, size, hipMemcpyDeviceToHost));

    CHECK(hipDeviceSynchronize());

    CHECK(hipHostFree(d_gradientX));
    CHECK(hipHostFree(d_gradientY));
    CHECK(hipHostFree(d_magnitude));
    CHECK(hipHostFree(d_direction));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end));
}


void cuda_compute_hog(unsigned char *magnitude, unsigned char *direction, int width, int height) {
    unsigned char *d_magnitude, *d_direction;
    float *d_bins;
    size_t size = width*height;
    int num_blocks = (width*height)/HOG_BLOCK_SIDE + 1;
    size_t nBytes = NUM_BINS*num_blocks*sizeof(float);

    struct Hog *hog = (struct Hog *)malloc(sizeof(struct Hog));
    hog->bins = (float *)calloc(NUM_BINS*num_blocks, sizeof(float));
    
    CHECK(hipHostMalloc((void **)&d_magnitude, size));
    CHECK(hipHostMalloc((void **)&d_direction, size));
    CHECK(hipHostMalloc((void **)&d_bins, nBytes));
    if(d_magnitude == NULL || d_direction == NULL || d_bins == NULL) {
        printf("Unable to allocate memory on GPU.\n");
        exit(EXIT_FAILURE);
    }

    // To do: implement streams
    CHECK(hipMemcpyAsync(d_magnitude, magnitude, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpyAsync(d_direction, direction, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpyAsync(d_bins, hog->bins, nBytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());

    dim3 block(HOG_BLOCK_SIDE, HOG_BLOCK_SIDE);
    dim3 grid((width + block.x - 1)/block.x, (height + block.y - 1)/block.y);


    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));
    float time;
    
    hipEventRecord(start, 0);
    hog_gpu<<< grid, block >>>(d_bins, d_magnitude, d_direction, width, height);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("GPU Elapsed time: %f sec\n\n", time/1000);

    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        printf("\n--> Error: %s\n", hipGetErrorString(err));
    }

    CHECK(hipMemcpyAsync(hog->bins, d_bins, nBytes, hipMemcpyDeviceToHost));

    CHECK(hipDeviceSynchronize());

    CHECK(hipHostFree(d_magnitude));
    CHECK(hipHostFree(d_direction));
    CHECK(hipHostFree(d_bins));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end));
}
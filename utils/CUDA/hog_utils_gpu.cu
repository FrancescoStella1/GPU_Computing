#include "hip/hip_runtime.h"
#include "../hog_utils.h"
#include "../timing.c"


__global__ void mag_dir_gpu(unsigned char *gradientX, unsigned char *gradientY, unsigned char *magnitude, unsigned char *direction, size_t size) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= size)
        return;
    
    float grad_x = gradientX[i];
    float grad_y = gradientY[i];

    float mag = sqrtf(powf(grad_x, 2) + powf(grad_y, 2));
    float atang = atan2f(grad_y, grad_x) * (180/PI);

    magnitude[i] = (unsigned char)mag;
    direction[i] = (unsigned char)atang;

}


__global__ void hog_gpu(float *bins, unsigned char *magnitude, unsigned char *direction, int width, int height) {
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    uint j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i >= width || j >= height)
        return;

    int lbin = direction[j*width + i]/DELTA_THETA;
    int ubin = lbin + 1;
    if(ubin>=NUM_BINS)
      ubin = 0;

    int cbin = (lbin + 0.5);

    unsigned char mag = magnitude[j*width + i];
    unsigned char dir = direction[j*width + i];
    float l_value =  mag * ((dir - (DELTA_THETA/2))/DELTA_THETA);
    float u_value = mag * ((dir - cbin)/DELTA_THETA);

    int blocks_per_row = (width + HOG_BLOCK_SIDE - 1)/HOG_BLOCK_SIDE;
    int block_idx = blockIdx.y * blocks_per_row + blockIdx.x;

    atomicAdd(&bins[block_idx*NUM_BINS + lbin], l_value);
    atomicAdd(&bins[block_idx*NUM_BINS + ubin], u_value);
}


void cuda_compute_mag_dir(unsigned char *gradientX, unsigned char *gradientY, unsigned char *magnitude, unsigned char *direction, int dim, char *log_file) {

    unsigned char *d_gradientX;
    unsigned char *d_gradientY;
    unsigned char *d_magnitude;
    unsigned char *d_direction;
    size_t size = dim;

    //memset(magnitude, 0, size);
    //memset(direction, 0, size);

    CHECK(hipHostMalloc((unsigned char **)&d_gradientX, size));
    CHECK(hipHostMalloc((unsigned char **)&d_gradientY, size));
    CHECK(hipHostMalloc((unsigned char **)&d_magnitude, size));
    CHECK(hipHostMalloc((unsigned char **)&d_direction, size));

    if(d_gradientX == NULL || d_gradientY == NULL || d_magnitude == NULL || d_direction == NULL)   {
        printf("Unable to allocate memory on GPU.\n");
        exit(EXIT_FAILURE);
    }

    CHECK(hipMemcpyAsync(d_gradientX, gradientX, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpyAsync(d_gradientY, gradientY, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpyAsync(d_magnitude, magnitude, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpyAsync(d_direction, direction, size, hipMemcpyHostToDevice));

    CHECK(hipDeviceSynchronize());

    dim3 block(HOG_BLOCK_SIDE);
    dim3 grid((size+block.x-1)/block.x);

    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));
    float time;
    
    hipEventRecord(start, 0);
    mag_dir_gpu<<< grid, block >>>(d_gradientX, d_gradientY, d_magnitude, d_direction, size);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("[Magnitude & Direction] - GPU Elapsed time: %f sec\n\n", time/1000);
    write_to_file(log_file, "Magnitude and Direction", time/1000, 1, 0);

    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        printf("\n--> Error: %s\n", hipGetErrorString(err));
    }

    CHECK(hipMemcpyAsync(magnitude, d_magnitude, size, hipMemcpyDeviceToHost));
    CHECK(hipMemcpyAsync(direction, d_direction, size, hipMemcpyDeviceToHost));

    CHECK(hipDeviceSynchronize());

    CHECK(hipHostFree(d_gradientX));
    CHECK(hipHostFree(d_gradientY));
    CHECK(hipHostFree(d_magnitude));
    CHECK(hipHostFree(d_direction));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end));
}


void cuda_compute_hog(float *hog, unsigned char *magnitude, unsigned char *direction, int width, int height, char *log_file) {
    unsigned char *d_magnitude, *d_direction;
    float *d_bins;
    size_t size = width*height;
    int num_blocks = (size + HOG_BLOCK_SIDE - 1)/HOG_BLOCK_SIDE;
    size_t nBytes = NUM_BINS*num_blocks*sizeof(float);
    hog = allocate_histograms(num_blocks);
    
    CHECK(hipHostMalloc((unsigned char **)&d_magnitude, size));
    CHECK(hipHostMalloc((unsigned char **)&d_direction, size));
    CHECK(hipHostMalloc((float **)&d_bins, nBytes));
    if(d_magnitude == NULL || d_direction == NULL || d_bins == NULL) {
        printf("Unable to allocate memory on GPU.\n");
        exit(EXIT_FAILURE);
    }

    // To do: implement streams
    CHECK(hipMemcpyAsync(d_magnitude, magnitude, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpyAsync(d_direction, direction, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpyAsync(d_bins, hog, nBytes, hipMemcpyHostToDevice));
    CHECK(hipDeviceSynchronize());

    dim3 block(HOG_BLOCK_SIDE, HOG_BLOCK_SIDE);
    dim3 grid((width + block.x - 1)/block.x, (height + block.y - 1)/block.y);


    hipEvent_t start, end;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&end));
    float time;
    
    hipEventRecord(start, 0);
    hog_gpu<<< grid, block >>>(d_bins, d_magnitude, d_direction, width, height);
    CHECK(hipDeviceSynchronize());
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, start, end);
    printf("[HOG Computation] - GPU Elapsed time: %f sec\n\n", time/1000);
    write_to_file(log_file, "HOG computation", time/1000, 1, 1);

    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        printf("\n--> Error: %s\n", hipGetErrorString(err));
    }

    CHECK(hipMemcpyAsync(hog, d_bins, nBytes, hipMemcpyDeviceToHost));

    CHECK(hipDeviceSynchronize());

    CHECK(hipHostFree(d_magnitude));
    CHECK(hipHostFree(d_direction));
    CHECK(hipHostFree(d_bins));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(end));
}